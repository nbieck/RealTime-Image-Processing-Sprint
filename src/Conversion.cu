#include "hip/hip_runtime.h"
#include "Conversion.h"

#include "GpuUtils.h"

// workarounds for incorrect highlighting
#include <>
#ifdef __INTELLISENSE__
	#define KERNEL_LAUNCH(GRID, BLOCK)
#else
	#define KERNEL_LAUNCH(GRID, BLOCK) <<<GRID, BLOCK>>>
#endif

#include "hip/hip_math_constants.h"

__device__ int idx2D(int x, int y, int width)
{
	return y * width + x;
}

__device__ float2 operator+(float2 a, float2 b)
{
	return make_float2(a.x + b.x, a.y + b.y);
}

__device__ float3 operator/(float3 a, float b)
{
	return make_float3(a.x / b, a.y / b, a.z / b);
}

__device__ float3 cartesianToSpherical(float3 cartesian)
{
	float r = sqrtf(cartesian.x * cartesian.x + cartesian.y * cartesian.y + cartesian.z * cartesian.z);

	float theta = atan2f(cartesian.y, cartesian.x);
	float phi = acosf(cartesian.z / r);

	return make_float3(theta, phi, r);
}

__device__ float3 sphericalToCartesian(float3 spherical)
{
	float z = cos(spherical.y) * spherical.z;
	float x = cos(spherical.x) * sin(spherical.y) * spherical.z;
	float y = sin(spherical.x) * sin(spherical.y) * spherical.z;

	return make_float3(x, y, z);
}

__device__ float3 cubeCoordFromXY(int x, int y, int squareDim)
{
	float3 cartCoord = make_float3(1,0,0);

	// identify cube face
	if (x < squareDim)
	{
		if (y < squareDim)
		{
			// -Y
			cartCoord = make_float3(
				x / (float)squareDim - 0.5,
				-0.5,
				-(y / (float)squareDim - 0.5)
			);
		}
		else
		{
			y -= squareDim;
			// -X
			cartCoord = make_float3(
				-0.5,
				-(x / (float)squareDim - 0.5),
				-(y / (float)squareDim - 0.5)
			);
		}
	}
	else if (x < 2 * squareDim)
	{
		x -= squareDim;
		if (y < squareDim)
		{
			// +X
			cartCoord = make_float3(
				0.5,
				x / (float)squareDim - 0.5,
				-(y / (float)squareDim - 0.5)
			);
		}
		else
		{
			y -= squareDim;
			// -Z
			cartCoord = make_float3(
				y / (float)squareDim - 0.5,
				-(x / (float)squareDim - 0.5),
				-0.5
			);
		}
	}
	else
	{
		x -= 2 * squareDim;
		if (y < squareDim)
		{
			// +Y
			cartCoord = make_float3(
				-(x / (float)squareDim - 0.5),
				0.5,
				-(y / (float)squareDim - 0.5)
			);
		}
		else
		{
			y -= squareDim;
			// +Z 
			cartCoord = make_float3(
				y / (float)squareDim - 0.5,
				x / (float)squareDim - 0.5,
				0.5
			);
		}
	}

	return cartCoord;
}

__device__ int argmax(float3 v)
{
	if (v.x > v.y)
	{
		if (v.x > v.z)
		{
			return 0;
		}
		return 2;
	}
	else
	{
		if (v.y > v.z)
		{
			return 1;
		}
		return 2;
	}
}

__device__ float3 abs(float3 v)
{
	return make_float3(abs(v.x), abs(v.y), abs(v.z));
}

__device__ float max(float3 v)
{
	return max(v.x, max(v.y, v.z));
}

__device__ float lookup(float3 v, int idx)
{
	switch (idx)
	{
	case 0:
		return v.x;
	case 1:
		return v.y;
	default:
		return v.z;
	}
}

enum class CubeFace
{
	PlusX,
	MinX,
	PlusY,
	MinY,
	PlusZ,
	MinZ
};

__device__ CubeFace getCubeFace(float3 coord)
{
	int max_axis = argmax(abs(coord));
	float max_val = lookup(coord, max_axis);

	switch (max_axis)
	{
	case 0:
		if (max_val > 0)
		{
			return CubeFace::PlusX;
		}
		else
		{
			return CubeFace::MinX;
		}
		break;
	case 1:
		if (max_val > 0)
		{
			return CubeFace::PlusY;
		}
		else
		{
			return CubeFace::MinY;
		}
		break;
	case 2:
		if (max_val > 0)
		{
			return CubeFace::PlusZ;
		}
		else
		{
			return CubeFace::MinZ;
		}
		break;
	default:
		return CubeFace::PlusX;
	}
}

__device__ float2 baseCoordFrom3D(float3 coord, CubeFace face)
{
	float2 result = make_float2(0, 0);

	coord = coord / (max(abs(coord)) * 2);

	switch (face)
	{
	case CubeFace::PlusX:
		result = make_float2(
			coord.y + 0.5,
			coord.z + 0.5
		);
		break;
	case CubeFace::MinX:
		result = make_float2(
			-coord.y + 0.5,
			coord.z + 0.5
		);
		break;
	case CubeFace::PlusY:
		result = make_float2(
			-coord.x + 0.5,
			coord.z + 0.5
		);
		break;
	case CubeFace::MinY:
		result = make_float2(
			coord.x + 0.5,
			coord.z + 0.5
		);
		break;
	case CubeFace::PlusZ:
		result = make_float2(
			coord.y + 0.5,
			-coord.x + 0.5
		);
		break;
	case CubeFace::MinZ:
		result = make_float2(
			coord.y + 0.5,
			coord.x + 0.5
		);
		break;
	}

	result.y = 1 - result.y;

	result.x /= 3.;
	result.y /= 2.;

	return result;
}

__device__ float2 offsetFromFace(CubeFace face)
{
	float2 result = make_float2(0, 0);

	switch (face)
	{
	case CubeFace::PlusX:
		result = make_float2(
			1./3.,
			0
		);
		break;
	case CubeFace::MinX:
		result = make_float2(
			0,
			0.5
		);
		break;
	case CubeFace::PlusY:
		result = make_float2(
			2./3.,
			0
		);
		break;
	case CubeFace::MinY:
		result = make_float2(
			0,
			0
		);
		break;
	case CubeFace::PlusZ:
		result = make_float2(
			2./3.,
			0.5
		);
		break;
	case CubeFace::MinZ:
		result = make_float2(
			1./3.,
			0.5
		);
		break;
	}

	return result;
}

// convert from a cartesian coordinate to the corresponsing spot on the cubemap texture
__device__ float2 cartesianToTexCoord(float3 cartesian)
{
	CubeFace face = getCubeFace(cartesian);

	float2 base_coord = baseCoordFrom3D(cartesian, face);
	float2 offset = offsetFromFace(face);

	return base_coord + offset;
}

__global__ void equiToCube(const uchar3* src, uchar3* dst, int out_width, int out_height, int in_width, int in_height)
{
	const int x = blockDim.x * blockIdx.x + threadIdx.x;
	const int y = blockDim.y * blockIdx.y + threadIdx.y;

	const int squareDim = out_height / 2;

	if (x < out_width && y < out_height)
	{
		float3 cubeCoord = cubeCoordFromXY(x, y, squareDim);

		float3 spherical = cartesianToSpherical(cubeCoord);

		float theta_norm = spherical.x / (HIP_PI * 2);
		if (theta_norm < 0)
		{
			theta_norm += 1;
		}
		float phi_norm = spherical.y / HIP_PI;

		dst[idx2D(x, y, out_width)] = src[idx2D((int)(theta_norm * in_width), (int)(phi_norm * in_height), in_width)];
	}
}

__global__ void cubeToEqui(const uchar3* src, uchar3* dst, int out_width, int out_height, int in_width, int in_height)
{
	const int x = blockDim.x * blockIdx.x + threadIdx.x;
	const int y = blockDim.y * blockIdx.y + threadIdx.y;

	if (x < out_width && y < out_height)
	{
		float3 sphericalCoord = make_float3(((float)x / out_width) * HIP_PI * 2, ((float)y / out_height) * HIP_PI, 2);
		float3 cartesian = sphericalToCartesian(sphericalCoord);

		float2 texCoord = cartesianToTexCoord(cartesian);

		dst[idx2D(x, y, out_width)] = src[idx2D((int)(texCoord.x * in_width), (int)(texCoord.y * in_height), in_width)];
	}
}

int divUp(int a, int b)
{
	return (a + b - 1) / b;
}

Image convertEquirectangularToCubemap(Image&& input)
{
	int squareDims = input.width() / 4;

	Image result(squareDims * 3, squareDims * 2);

	GpuBuffer<uchar3> d_input(input.width() * input.height());
	GpuBuffer<uchar3> d_output(result.width() * result.height());

	d_input.upload(input.data());
	
	dim3 block(16, 16);
	dim3 grid(divUp(result.width(), block.x), divUp(result.height(), block.y));

	equiToCube KERNEL_LAUNCH(grid, block) (d_input.ptr(), d_output.ptr(), result.width(), result.height(), input.width(), input.height());

	d_output.download(result.data());

	return result;
}

Image convertCubemapToEquirectangular(Image&& input)
{
	int squareDims = input.width() / 3;

	Image result(squareDims * 4, squareDims * 2);

	GpuBuffer<uchar3> d_input(input.width() * input.height());
	GpuBuffer<uchar3> d_output(result.width() * result.height());

	d_input.upload(input.data());
	
	dim3 block(16, 16);
	dim3 grid(divUp(result.width(), block.x), divUp(result.height(), block.y));

	cubeToEqui KERNEL_LAUNCH(grid, block) (d_input.ptr(), d_output.ptr(), result.width(), result.height(), input.width(), input.height());

	d_output.download(result.data());

	return result;
}